/*
 * This file is part of Xpra.
 * Copyright (C) 2021 Antoine Martin <antoine@xpra.org>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void RGBX_to_RGB(int src_w, int src_h,
                             int srcPitch, uint8_t *srcImage,
                             int dst_w, int dst_h,
                             int dstPitch, uint8_t *dstImage)
{
    const uint32_t gx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t gy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t src_x = gx * src_w / dst_w;
    const uint32_t src_y = gy * src_h / dst_h;

    if ((src_x < src_w) & (src_y < src_h) & (gx < dst_w) & (gy < dst_h)) {
        uint32_t si = (src_y * srcPitch) + src_x * 4;
        uint32_t di = (gy * dstPitch) + gx*3;
        //A = srcImage[si+3];
        dstImage[di]   = srcImage[si+2];
        dstImage[di+1] = srcImage[si+1];
        dstImage[di+2] = srcImage[si];
    }
}
