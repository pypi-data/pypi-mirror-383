/*
 * This file is part of Xpra.
 * Copyright (C) 2013-2024 Antoine Martin <antoine@xpra.org>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

// Y = 0.257 * R + 0.504 * G + 0.098 * B + 0
#define YR 0.257
#define YG 0.504
#define YB 0.098
#define YC 0
// U = -0.148 * R - 0.291 * G + 0.439 * B + 128
#define UR -0.148
#define UG -0.291
#define UB 0.439
#define UC 128
// V = 0.439 * R - 0.368 * G - 0.071 * B + 128
#define VR 0.439
#define VG -0.368
#define VB -0.071
#define VC 128

extern "C" __global__ void XRGB_to_NV12(uint8_t *srcImage, int src_w, int src_h, int srcPitch,
                          uint8_t *dstImage, int dst_w, int dst_h, int dstPitch,
                          int w, int h)
{
    const uint32_t gx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t gy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t src_y = gy*2 * src_h / dst_h;
    const uint32_t src_x = gx*2 * src_w / dst_w;

    if ((src_x < w) & (src_y < h)) {
        //4 bytes per pixel, and 2 pixels width/height at a time:
        //byte index:
        uint32_t si = (src_y * srcPitch) + src_x * 4;

        //we may read up to 4 32-bit RGB pixels:
        uint8_t R[4];
        uint8_t G[4];
        uint8_t B[4];
        uint8_t j = 0;
        R[0] = srcImage[si+1];
        G[0] = srcImage[si+2];
        B[0] = srcImage[si+3];
        for (j=1; j<4; j++) {
            R[j] = R[0];
            G[j] = G[0];
            B[j] = B[0];
        }

        //write up to 4 Y pixels:
        uint32_t di = (gy * 2 * dstPitch) + gx * 2;
        dstImage[di] = __float2int_rn(YR * R[0] + YG * G[0] + YB * B[0] + YC);
        if (gx*2 + 1 < src_w) {
            R[1] = srcImage[si+5];
            G[1] = srcImage[si+6];
            B[1] = srcImage[si+7];
            dstImage[di + 1] = __float2int_rn(YR * R[1] + YG * G[1] + YB * B[1] + YC);
        }
        if (gy*2 + 1 < src_h) {
            si += srcPitch;
            di += dstPitch;
            R[2] = srcImage[si+1];
            G[2] = srcImage[si+2];
            B[2] = srcImage[si+3];
            dstImage[di] = __float2int_rn(YR * R[2] + YG * G[2] + YB * B[2] + YC);
            if (gx*2 + 1 < src_w) {
                R[3] = srcImage[si+5];
                G[3] = srcImage[si+6];
                B[3] = srcImage[si+7];
                dstImage[di + 1] = __float2int_rn(YR * R[3] + YG * G[3] + YB * B[3] + YC);
            }
        }

        //write 1 U and 1 V pixel:
        float u = 0;
        float v = 0;
        for (j=0; j<4; j++) {
            u += UR * R[j] + UG * G[j] + UB * B[j] + UC;
            v += VR * R[j] + VG * G[j] + VB * B[j] + VC;
        }
        di = (dst_h + gy) * dstPitch + gx * 2;
        dstImage[di]      = __float2int_rn(u / 4.0);
        dstImage[di + 1]  = __float2int_rn(v / 4.0);
    }
}
