/*
 * This file is part of Xpra.
 * Copyright (C) 2013-2024 Antoine Martin <antoine@xpra.org>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

// Y = 0.299 * R + 0.587 * G + 0.114 * B + 0
#define YR 0.299
#define YG 0.587
#define YB 0.114
#define YC 0
// U = -0.168736 * R - 0.331264 * G + 0.5 * B + 128
#define UR -0.168736
#define UG -0.331264
#define UB 0.5
#define UC 128
// V = 0.5 * R - 0.418688 * G - 0.081312 * B + 128
#define VR 0.5
#define VG -0.418688
#define VB -0.081312
#define VC 128

extern "C" __global__ void BGRX_to_NV12(uint8_t *srcImage, int src_w, int src_h, int srcPitch,
                          uint8_t *dstImage, int dst_w, int dst_h, int dstPitch,
                          int w, int h)
{
    const uint32_t gx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t gy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t src_y = gy*2 * src_h / dst_h;
    const uint32_t src_x = gx*2 * src_w / dst_w;

    if ((src_x < w) & (src_y < h)) {
        //4 bytes per pixel, and 2 pixels width/height at a time:
        //byte index:
        uint32_t si = (src_y * srcPitch) + src_x * 4;

        //we may read up to 4 32-bit RGB pixels:
        uint8_t R[4];
        uint8_t G[4];
        uint8_t B[4];
        uint8_t j = 0;
        R[0] = srcImage[si+2];
        G[0] = srcImage[si+1];
        B[0] = srcImage[si];
        for (j=1; j<4; j++) {
            R[j] = R[0];
            G[j] = G[0];
            B[j] = B[0];
        }

        //write up to 4 Y pixels:
        uint32_t di = (gy * 2 * dstPitch) + gx * 2;
        dstImage[di] = __float2int_rn(YR * R[0] + YG * G[0] + YB * B[0] + YC);
        if (gx*2 + 1 < src_w) {
            R[1] = srcImage[si+6];
            G[1] = srcImage[si+5];
            B[1] = srcImage[si+4];
            dstImage[di + 1] = __float2int_rn(YR * R[1] + YG * G[1] + YB * B[1] + YC);
        }
        if (gy*2 + 1 < src_h) {
            si += srcPitch;
            di += dstPitch;
            R[2] = srcImage[si+2];
            G[2] = srcImage[si+1];
            B[2] = srcImage[si];
            dstImage[di] = __float2int_rn(YR * R[2] + YG * G[2] + YB * B[2] + YC);
            if (gx*2 + 1 < src_w) {
                R[3] = srcImage[si+6];
                G[3] = srcImage[si+5];
                B[3] = srcImage[si+4];
                dstImage[di + 1] = __float2int_rn(YR * R[3] + YG * G[3] + YB * B[3] + YC);
            }
        }

        //write 1 U and 1 V pixel:
        float u = 0;
        float v = 0;
        for (j=0; j<4; j++) {
            u += UR * R[j] + UG * G[j] + UB * B[j] + UC;
            v += VR * R[j] + VG * G[j] + VB * B[j] + VC;
        }
        di = (dst_h + gy) * dstPitch + gx * 2;
        dstImage[di]      = __float2int_rn(u / 4.0);
        dstImage[di + 1]  = __float2int_rn(v / 4.0);
    }
}
