/*
 * This file is part of Xpra.
 * Copyright (C) 2013-2024 Antoine Martin <antoine@xpra.org>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

// Y = 0.299 * R + 0.587 * G + 0.114 * B + 0
#define YR 0.299
#define YG 0.587
#define YB 0.114
#define YC 0
// U = -0.168736 * R - 0.331264 * G + 0.5 * B + 128
#define UR -0.168736
#define UG -0.331264
#define UB 0.5
#define UC 128
// V = 0.5 * R - 0.418688 * G - 0.081312 * B + 128
#define VR 0.5
#define VG -0.418688
#define VB -0.081312
#define VC 128

extern "C" __global__ void XRGB_to_YUV444(uint8_t *srcImage, int src_w, int src_h, int srcPitch,
                             uint8_t *dstImage, int dst_w, int dst_h, int dstPitch,
                             int w, int h)
{
    const uint32_t gx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t gy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t src_y = gy * src_h / dst_h;
    const uint32_t src_x = gx * src_w / dst_w;

    if ((src_x < w) & (src_y < h)) {
        uint8_t R;
        uint8_t G;
        uint8_t B;
        //one 32-bit RGB pixel at a time:
        uint32_t si = (src_y * srcPitch) + src_x * 4;
        R = srcImage[si+1];
        G = srcImage[si+2];
        B = srcImage[si+3];

        uint32_t di;
        di = (gy * dstPitch) + gx;
        dstImage[di] = __float2int_rn(YR * R + YG * G + YB * B + YC);
        di += dstPitch*dst_h;
        dstImage[di] = __float2int_rn(UR * R + UG * G + UB * B + UC);
        di += dstPitch*dst_h;
        dstImage[di] = __float2int_rn(VR * R + VG * G + VB * B + VC);
    }
}
