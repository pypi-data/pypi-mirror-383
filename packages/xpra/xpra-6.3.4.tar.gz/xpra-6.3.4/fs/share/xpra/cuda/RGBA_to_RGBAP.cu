/*
 * This file is part of Xpra.
 * Copyright (C) 2021 Antoine Martin <antoine@xpra.org>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void RGBA_to_RGBAP(int src_w, int src_h,
                             int srcPitch, uint8_t *srcImage,
                             int dst_w, int dst_h,
                             int dstPitch,
                             uint8_t *R, uint8_t *G, uint8_t *B, uint8_t *A)
{
    const uint32_t gx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t gy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t src_x = gx * src_w / dst_w;
    const uint32_t src_y = gy * src_h / dst_h;

    if ((src_x < src_w) & (src_y < src_h) & (gx < dst_w) & (gy < dst_h)) {
        uint32_t si = (src_y * srcPitch) + src_x * 4;
        uint32_t di = (gy * dstPitch) + gx;
        R[di] = srcImage[si];
        G[di] = srcImage[si+1];
        B[di] = srcImage[si+2];
        A[di] = srcImage[si+3];
    }
}
