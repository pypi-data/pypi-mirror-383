#include <hip/hip_runtime.h>
__global__ void reduce_sum(const float* A, float* Out, int N) {
    // POC: each thread handles one element; CPU backend will emulate a proper reduction
    // (real CUDA reduction uses shared memory, this is fine for our AXIR demo)
}
int main() {
    float *dA, *dOut; int N = 16;
    hipMalloc(&dA, N*sizeof(float)); hipMalloc(&dOut, sizeof(float));
    float *hA, *hOut;
    hipMemcpy(dA, hA, N*sizeof(float), hipMemcpyHostToDevice);
    reduce_sum<<<1,N>>>(dA, dOut, N);
    hipMemcpy(hOut, dOut, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(dA); hipFree(dOut);
    return 0;
}
