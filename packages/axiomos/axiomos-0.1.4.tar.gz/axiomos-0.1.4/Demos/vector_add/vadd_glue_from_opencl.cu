#include <hip/hip_runtime.h>
// AXIR -> CUDA (glue POC)

// DeviceSelect: auto
// kernel k -> vector_add
hipMalloc(&dA, N * sizeof(float));
hipMalloc(&dB, N * sizeof(float));
hipMalloc(&dC, N * sizeof(float));
hipMemcpy(dA, hA, N*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(dB, hB, N*sizeof(float), hipMemcpyHostToDevice);
// setarg k[0]=dA
// setarg k[1]=dB
// setarg k[2]=dC
// setarg k[3]=N
vector_add<<<&global,&local,0,0>>>(dA, dB, dC, N);
hipMemcpy(hC, dC, N*sizeof(float), hipMemcpyDeviceToHost);
hipDeviceSynchronize();
hipFree(dA);
hipFree(dB);
hipFree(dC);
