#include "hip/hip_runtime.h"
// SPDX-License-Identifier: GPL-3.0-or-later
/*
    NepTrainKit NEP descriptor kernels (extracted/adapted)
    Copyright (C) 2025 NepTrainKit contributors

    This file contains kernels derived from GPUMD
    (https://github.com/brucefan1983/GPUMD) by Zheyong Fan and the
    GPUMD development team, which is licensed under the GNU General
    Public License version 3 (or later).

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "nep_desc.cuh"

// This file duplicates a subset of NEP kernels to compute descriptors only.

__global__ void gpu_find_neighbor_list_desc(
  const NEP::ParaMB paramb,
  const int N,
  const int* Na,
  const int* Na_sum,
  const bool use_typewise_cutoff,
  const int* g_type,
  const float g_rc_radial,
  const float g_rc_angular,
  const float* __restrict__ g_box,
  const float* __restrict__ g_box_original,
  const int* __restrict__ g_num_cell,
  const float* x,
  const float* y,
  const float* z,
  int* NN_radial,
  int* NL_radial,
  int* NN_angular,
  int* NL_angular,
  float* x12_radial,
  float* y12_radial,
  float* z12_radial,
  float* x12_angular,
  float* y12_angular,
  float* z12_angular)
{
  int N1 = Na_sum[blockIdx.x];
  int N2 = N1 + Na[blockIdx.x];
  for (int n1 = N1 + threadIdx.x; n1 < N2; n1 += blockDim.x) {
    const float* __restrict__ box = g_box + 18 * blockIdx.x;
    const float* __restrict__ box_original = g_box_original + 9 * blockIdx.x;
    const int* __restrict__ num_cell = g_num_cell + 3 * blockIdx.x;
    float x1 = x[n1];
    float y1 = y[n1];
    float z1 = z[n1];
    int t1 = g_type[n1];
    int count_radial = 0;
    int count_angular = 0;
    for (int n2 = N1; n2 < N2; ++n2) {
      for (int ia = 0; ia < num_cell[0]; ++ia) {
        for (int ib = 0; ib < num_cell[1]; ++ib) {
          for (int ic = 0; ic < num_cell[2]; ++ic) {
            if (ia == 0 && ib == 0 && ic == 0 && n1 == n2) {
              continue; // exclude self
            }
            float delta_x = box_original[0] * ia + box_original[1] * ib + box_original[2] * ic;
            float delta_y = box_original[3] * ia + box_original[4] * ib + box_original[5] * ic;
            float delta_z = box_original[6] * ia + box_original[7] * ib + box_original[8] * ic;
            float x12 = x[n2] + delta_x - x1;
            float y12 = y[n2] + delta_y - y1;
            float z12 = z[n2] + delta_z - z1;
            dev_apply_mic(box, x12, y12, z12);
            float distance_square = x12 * x12 + y12 * y12 + z12 * z12;
            int t2 = g_type[n2];
            float rc_radial = g_rc_radial;
            float rc_angular = g_rc_angular;
            if (use_typewise_cutoff) {
              int z1 = paramb.atomic_numbers[t1];
              int z2 = paramb.atomic_numbers[t2];
              rc_radial = min((COVALENT_RADIUS[z1] + COVALENT_RADIUS[z2]) * paramb.typewise_cutoff_radial_factor, rc_radial);
              rc_angular = min((COVALENT_RADIUS[z1] + COVALENT_RADIUS[z2]) * paramb.typewise_cutoff_angular_factor, rc_angular);
            }
            if (distance_square < rc_radial * rc_radial) {
              NL_radial[count_radial * N + n1] = n2;
              x12_radial[count_radial * N + n1] = x12;
              y12_radial[count_radial * N + n1] = y12;
              z12_radial[count_radial * N + n1] = z12;
              count_radial++;
            }
            if (distance_square < rc_angular * rc_angular) {
              NL_angular[count_angular * N + n1] = n2;
              x12_angular[count_angular * N + n1] = x12;
              y12_angular[count_angular * N + n1] = y12;
              z12_angular[count_angular * N + n1] = z12;
              count_angular++;
            }
          }
        }
      }
    }
    NN_radial[n1] = count_radial;
    NN_angular[n1] = count_angular;
  }
}

__global__ void find_descriptors_radial_desc(
  const int N,
  const int* g_NN,
  const int* g_NL,
  const NEP::ParaMB paramb,
  const NEP::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_descriptors)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    int t1 = g_type[n1];
    int neighbor_number = g_NN[n1];
    float q[MAX_NUM_N] = {0.0f};
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = n1 + N * i1;
      int n2 = g_NL[index];
      float x12 = g_x12[index];
      float y12 = g_y12[index];
      float z12 = g_z12[index];
      float d12 = sqrtf(x12 * x12 + y12 * y12 + z12 * z12);
      float fc12;
      int t2 = g_type[n2];
      float rc = paramb.rc_radial;
      if (paramb.use_typewise_cutoff) {
        rc = min((COVALENT_RADIUS[paramb.atomic_numbers[t1]] + COVALENT_RADIUS[paramb.atomic_numbers[t2]]) * paramb.typewise_cutoff_radial_factor, rc);
      }
      float rcinv = 1.0f / rc;
      find_fc(rc, rcinv, d12, fc12);

      float fn12[MAX_NUM_N];
      find_fn(paramb.basis_size_radial, rcinv, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_radial; ++k) {
          int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        q[n] += gn12;
      }
    }
    for (int n = 0; n <= paramb.n_max_radial; ++n) {
      g_descriptors[n1 + n * N] = q[n];
    }
  }
}

__global__ void find_descriptors_angular_desc(
  const int N,
  const int* g_NN,
  const int* g_NL,
  const NEP::ParaMB paramb,
  const NEP::ANN annmb,
  const int* __restrict__ g_type,
  const float* __restrict__ g_x12,
  const float* __restrict__ g_y12,
  const float* __restrict__ g_z12,
  float* g_descriptors,
  float* g_sum_fxyz)
{
  int n1 = threadIdx.x + blockIdx.x * blockDim.x;
  if (n1 < N) {
    int t1 = g_type[n1];
    int neighbor_number = g_NN[n1];
    float q[MAX_DIM_ANGULAR] = {0.0f};

    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < neighbor_number; ++i1) {
        int index = n1 + N * i1;
        int n2 = g_NL[n1 + N * i1];
        float x12 = g_x12[index];
        float y12 = g_y12[index];
        float z12 = g_z12[index];
        float d12 = sqrtf(x12 * x12 + y12 * y12 + z12 * z12);
        float fc12;
        int t2 = g_type[n2];
        float rc = paramb.rc_angular;
        if (paramb.use_typewise_cutoff) {
          rc = min((COVALENT_RADIUS[paramb.atomic_numbers[t1]] + COVALENT_RADIUS[paramb.atomic_numbers[t2]]) * paramb.typewise_cutoff_angular_factor, rc);
        }
        float rcinv = 1.0f / rc;
        find_fc(rc, rcinv, d12, fc12);
        float fn12[MAX_NUM_N];
        find_fn(paramb.basis_size_angular, rcinv, d12, fc12, fn12);
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_angular; ++k) {
          int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        accumulate_s(paramb.L_max, d12, x12, y12, z12, gn12, s);
      }
      find_q(paramb.L_max, paramb.num_L, paramb.n_max_angular + 1, n, s, q);
      for (int abc = 0; abc < (paramb.L_max + 1) * (paramb.L_max + 1) - 1; ++abc) {
        g_sum_fxyz[(n * ((paramb.L_max + 1) * (paramb.L_max + 1) - 1) + abc) * N + n1] = s[abc];
      }
    }

    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      for (int l = 0; l < paramb.num_L; ++l) {
        int ln = l * (paramb.n_max_angular + 1) + n;
        g_descriptors[n1 + ((paramb.n_max_radial + 1) + ln) * N] = q[ln];
      }
    }
  }
}

